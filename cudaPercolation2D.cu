#include "hip/hip_runtime.h"
// #include <pycuda-complex.hpp>
// #include <surface_functions.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

typedef unsigned char uchar;

texture< int, hipTextureType2D, hipReadModeElementType> tex_isFree;
texture<float, hipTextureType2D, hipReadModeElementType> tex_concentrationIn;
// surface< void, cudaSurfaceType2D> surf_concentrationOut;

// __global__ void countFreeNeighbors_kernel( const int nWidth, const int nHeight, int *nFreeAll){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int tid = t_j + t_i*blockDim.x*gridDim.x;
//   
//   uchar left   = tex2D( tex_isFree, t_j-1, t_i );
//   uchar right  = tex2D( tex_isFree, t_j+1, t_i );
//   uchar up     = tex2D( tex_isFree, t_j, t_i+1 );
//   uchar down   = tex2D( tex_isFree, t_j, t_i-1 );
//   
//   //Set PERIODIC boundary conditions
//   if (t_i == 0)           down = tex2D( tex_isFree, t_j, nHeight-1 );
//   if (t_i == (nHeight-1))   up = tex2D( tex_isFree, t_j, 0 );
//   if (t_j == 0)           left = tex2D( tex_isFree, nWidth-1, t_i );
//   if (t_j == (nWidth-1)) right = tex2D( tex_isFree, 0, t_i );
//   
//   int nFree = 0;
//   if ( left )  nFree += 1;
//   if ( right ) nFree += 1;
//   if ( down )  nFree += 1;
//   if ( up )    nFree += 1;
// 
//   nFreeAll[tid] = nFree;
// }
/////////////////////////////////////////////////////////////////////////////////////// 
/////////////////////////////////////////////////////////////////////////////////////// 
__global__ void main_kernel_tex( const int nWidth, const int nHeight, float hx, int *isFreeAll,
			     float *concentrationOut ){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int tid = t_j + t_i*blockDim.x*gridDim.x;
   
  //Read neighbors occupancy
  int left_isFree   =  tex2D( tex_isFree, t_j-1, t_i ); ; 
  int right_isFree  =  tex2D( tex_isFree, t_j+1, t_i ); ; 
  int up_isFree     =  tex2D( tex_isFree, t_j, t_i+1 ); ; 
  int down_isFree   =  tex2D( tex_isFree, t_j, t_i-1 ); ; 
  //Set PERIODIC boundary conditions
  if (t_i == 0)           down_isFree = isFreeAll[ t_j + (nHeight-1)*nWidth ];
  if (t_i == (nHeight-1))   up_isFree = isFreeAll[ t_j ];
  if (t_j == 0)           left_isFree = isFreeAll[ (nWidth-1) + (t_i)*nWidth ];
  if (t_j == (nWidth-1)) right_isFree = isFreeAll[ (t_i)*nWidth ];

  //Read neighbors concentration
  float center_C = tex2D( tex_concentrationIn, t_j,   t_i );
  float left_C   = tex2D( tex_concentrationIn, t_j-1, t_i );
  float right_C  = tex2D( tex_concentrationIn, t_j+1, t_i );
  float up_C     = tex2D( tex_concentrationIn, t_j, t_i+1 );
  float down_C   = tex2D( tex_concentrationIn, t_j, t_i-1 );
  //Set PERIODIC boundary conditions
  if (t_i == 0)           down_C = tex2D( tex_concentrationIn, t_j, nHeight-1 );
  if (t_i == (nHeight-1))   up_C = tex2D( tex_concentrationIn, t_j, 0 );
  if (t_j == 0)           left_C = tex2D( tex_concentrationIn, nWidth-1, t_i );
  if (t_j == (nWidth-1)) right_C = tex2D( tex_concentrationIn, 0, t_i );
 
  float newConcentration = 0.25f*( left_C + right_C + down_C + up_C ) +
         0.25f*( 4 - ( left_isFree + right_isFree + down_isFree + up_isFree ) )*center_C;

//   float newConcentration = hx*left_C + (1.f - hx)*(right_C + down_C + up_C )/3.f +
//       ( hx*(1 - right_isFree) + (1.f-hx)*( 3 - ( left_isFree + down_isFree + up_isFree ) )/3.f )*center_C;
//       
  if ( isFreeAll[tid] )  concentrationOut[tid] = newConcentration;
}




/////////////////////////////////////////////////////////////////////////////////////// 
/////////////////////////////////////////////////////////////////////////////////////// 
__global__ void main_kernel_shared( const int nWidth, const int nHeight, cudaP hx, int *isFreeAll,
			          cudaP *concIn, cudaP *concentrationOut ){
  const int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  const int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  const int tid = t_j + t_i*blockDim.x*gridDim.x;
  
  //Read my neighbors concentration
  __shared__ int   isFree_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  __shared__ cudaP concIn_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  concIn_sh[threadIdx.x+1][threadIdx.y+1] =    concIn[tid] ;
  isFree_sh[threadIdx.x+1][threadIdx.y+1] = isFreeAll[tid];
  //Left boundary
  if (t_j == 0){
    concIn_sh[0][threadIdx.y+1] =    concIn[ (nWidth-1) + t_i*nWidth ];
    isFree_sh[0][threadIdx.y+1] = isFreeAll[ (nWidth-1) + t_i*nWidth ];
  }
  else if ( threadIdx.x == 0 ){
    concIn_sh[0][threadIdx.y+1] =    concIn[ (t_j-1) + t_i*nWidth ];
    isFree_sh[0][threadIdx.y+1] = isFreeAll[ (t_j-1) + t_i*nWidth ];
  }
  //Right boundary
  if (t_j == nWidth-1){
    concIn_sh[blockDim.x+1][threadIdx.y+1] =    concIn[ t_i*nWidth ];
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFreeAll[ t_i*nWidth ];
  }
  else if ( threadIdx.x == blockDim.x-1 ){
    concIn_sh[blockDim.x+1][threadIdx.y+1] =    concIn[ (t_j+1) + t_i*nWidth ];
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFreeAll[ (t_j+1) + t_i*nWidth ];
  }
  //Down boundary
  if (t_i == 0){
    concIn_sh[threadIdx.x+1][0] =    concIn[ t_j + (nHeight-1)*nWidth ];
    isFree_sh[threadIdx.x+1][0] = isFreeAll[ t_j + (nHeight-1)*nWidth ];
  }
  else if ( threadIdx.y == 0 ){
    concIn_sh[threadIdx.x+1][0] =    concIn[ t_j + (t_i-1)*nWidth ];
    isFree_sh[threadIdx.x+1][0] = isFreeAll[ t_j + (t_i-1)*nWidth ];
  }
  //Up boundary
  if (t_i == nHeight-1){
    concIn_sh[threadIdx.x+1][blockDim.y+1] =    concIn[ t_j ];
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFreeAll[ t_j ];
  }
  else if ( threadIdx.y == blockDim.y-1 ){
    concIn_sh[threadIdx.x+1][blockDim.y+1] =    concIn[ t_j + (t_i+1)*nWidth ];
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFreeAll[ t_j + (t_i+1)*nWidth ];
  }
  __syncthreads();
  
  cudaP newConc = 0.25*( concIn_sh[threadIdx.x][threadIdx.y+1] + concIn_sh[threadIdx.x+2][threadIdx.y+1] +
                          concIn_sh[threadIdx.x+1][threadIdx.y] + concIn_sh[threadIdx.x+1][threadIdx.y+2] )+
         0.25*( 4 - ( isFree_sh[threadIdx.x][threadIdx.y+1] + isFree_sh[threadIdx.x+2][threadIdx.y+1] + 
                      isFree_sh[threadIdx.x+1][threadIdx.y] + isFree_sh[threadIdx.x+1][threadIdx.y+2] ) )*concIn_sh[threadIdx.x+1][threadIdx.y+1];
	 
	 
  
//   cudaP newConc = hx*concIn_sh[threadIdx.x][threadIdx.y+1] + 
//     (1 - hx)*( concIn_sh[threadIdx.x+2][threadIdx.y+1] + concIn_sh[threadIdx.x+1][threadIdx.y] + concIn_sh[threadIdx.x+1][threadIdx.y+2] )/3 +
//     ( hx*(1 - right_isFree) + (1 - hx)*( 3 - ( left_isFree + down_isFree + up_isFree ) )/3 )*concIn_sh[threadIdx.x+1][threadIdx.y+1];
	 
//   cudaP newConcentration = hx*left_C + (1.f - hx)*(right_C + down_C + up_C )/3.f +
//     hx*(1 - right_isFree)*center_C + (1.f-hx)*( 3 - ( left_isFree + down_isFree + up_isFree ) )/3.f*center_C;
 
  if ( isFree_sh[threadIdx.x+1][threadIdx.y+1] )  concentrationOut[tid] = newConc;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// __global__ void findActivity_kernel( cudaP minVal, cudaP *concentration, uchar *activeBlocks ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int tid = t_j + t_i*blockDim.x*gridDim.x;
//   int tid_b = threadIdx.x + threadIdx.y*blockDim.x;
// 
//   __shared__ cudaP concentration_sh[ %(THREADS_PER_BLOCK)s ];
//   concentration_sh[tid_b] = concentration[tid];
//   __syncthreads();
//   
//   int i = blockDim.x*blockDim.y / 2;
//   while ( i > 0 ){
//     if ( tid_b < i ) concentration_sh[tid_b] = concentration_sh[tid_b] + concentration_sh[tid_b+i];
//     __syncthreads();
//     i /= 2;
//   }
//   if ( tid_b == 0 ){
//     if (concentration_sh[0] >= minVal ) {
//       activeBlocks[ blockIdx.x + blockIdx.y*gridDim.x ] = (uchar) 1;
//       //right 
//       if (blockIdx.x < gridDim.x-1) activeBlocks[ (blockIdx.x+1) + blockIdx.y*gridDim.x ] = (uchar) 1;
//       //left
//       if (blockIdx.x > 0) activeBlocks[ (blockIdx.x-1) + blockIdx.y*gridDim.x ] = (uchar) 1;
//       //up 
//       if (blockIdx.y < gridDim.y-1) activeBlocks[ blockIdx.x + (blockIdx.y+1)*gridDim.x  ] = (uchar) 1;
//       //Down 
//       if (blockIdx.y > 0) activeBlocks[ blockIdx.x + (blockIdx.y-1)*gridDim.x  ] = (uchar) 1;
//     }
//   }
// }
// ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// __global__ void getActivity_kernel(  uchar *activeBlocks, uchar *activeThreads ){
//   int t_j = blockIdx.x*blockDim.x + threadIdx.x;
//   int t_i = blockIdx.y*blockDim.y + threadIdx.y;
//   int tid = t_j + t_i*blockDim.x*gridDim.x ;
//   int tid_b = threadIdx.x + threadIdx.y*blockDim.x;
//   int bid = blockIdx.x + blockIdx.y*gridDim.x;
//   
//   __shared__ uchar activeBlock;
//   if (tid_b == 0 ) activeBlock = activeBlocks[bid];
//   __syncthreads();
//   
//   if ( activeBlock ) activeThreads[tid] = (uchar) 1;
//   else activeThreads[tid] = (uchar) 0;
// }

/*
__global__ void main_kernel( const int nWidth, const int nHeight, int *nFreeAll, uchar *isFree, cudaP *concentrationIn, cudaP *concentrationOut ){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int tid = t_j + t_i*blockDim.x*gridDim.x;
  
  cudaP localConc = concentrationIn[tid];
  
  __shared__ uchar isFree_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  __shared__ cudaP concentrationIn_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  __shared__ cudaP concentrationOut_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  isFree_sh[threadIdx.x+1][threadIdx.y+1] = isFree[tid]; 
  concentrationIn_sh[threadIdx.x+1][threadIdx.y+1] = localConc ;
  concentrationOut_sh[threadIdx.x+1][threadIdx.y+1] = cudaP(0.);
  
  //Left boundary
  if (t_j == 0){  //Set periodic boundary
    isFree_sh[0][threadIdx.y+1] = isFree[ (nWidth-1) + t_i*nWidth ];
    concentrationIn_sh[0][threadIdx.y+1] = concentrationIn[ (nWidth-1) + t_i*nWidth ];
    concentrationOut_sh[0][threadIdx.y+1] = cudaP(0.);
  }
  else if ( threadIdx.x == 0 ){
    isFree_sh[0][threadIdx.y+1] = isFree[ (t_j-1) + t_i*nWidth ];
    concentrationIn_sh[0][threadIdx.y+1] = concentrationIn[ (t_j-1) + t_i*nWidth ];
    concentrationOut_sh[0][threadIdx.y+1] = cudaP(0.);
  }
  //Right boundary
  if (t_j == nWidth-1){  //Set periodic boundary
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFree[ t_i*nWidth ];
    concentrationIn_sh[blockDim.x+1][threadIdx.y+1] = concentrationIn[ t_i*nWidth ];
    concentrationOut_sh[blockDim.x+1][threadIdx.y+1] = cudaP(0.);
  }
  else if ( threadIdx.x == blockDim.x-1 ){
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFree[ (t_j+1) + t_i*nWidth ];
    concentrationIn_sh[blockDim.x+1][threadIdx.y+1] = concentrationIn[ (t_j+1) + t_i*nWidth ];
    concentrationOut_sh[blockDim.x+1][threadIdx.y+1] = cudaP(0.);
  }
  //Down boundary
  if (t_i == 0){  //Set periodic boundary
    isFree_sh[threadIdx.x+1][0] = isFree[ t_j + (nHeight-1)*nWidth ];
    concentrationIn_sh[threadIdx.x+1][0] = concentrationIn[ t_j + (nHeight-1)*nWidth ];
    concentrationOut_sh[threadIdx.x+1][0] = cudaP(0.);
  }
  else if ( threadIdx.y == 0 ){
    isFree_sh[threadIdx.x+1][0] = isFree[ t_j + (t_i-1)*nWidth ];
    concentrationIn_sh[threadIdx.x+1][0] = concentrationIn[ t_j + (t_i-1)*nWidth ];
    concentrationOut_sh[threadIdx.x+1][0] = cudaP(0.);
  }
  //Up boundary
  if (t_i == nHeight-1){  //Set periodic boundary
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFree[ t_j ];
    concentrationIn_sh[threadIdx.x+1][blockDim.y+1] = concentrationIn[ t_j ];
    concentrationOut_sh[threadIdx.x+1][blockDim.y+1] = cudaP(0.);
  }
  else if ( threadIdx.y == blockDim.y-1 ){
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFree[ t_j + (t_i+1)*nWidth ];
    concentrationIn_sh[threadIdx.x+1][blockDim.y+1] = concentrationIn[ t_j + (t_i+1)*nWidth ];
    concentrationOut_sh[threadIdx.x+1][blockDim.y+1] = cudaP(0.);
  }
  __syncthreads();
  
  int nFree = nFreeAll[tid];
//   concentrationOut[tid] = nFree;
  cudaP sendVal = localConc/nFree;
  if (isFree_sh[threadIdx.x][threadIdx.y + 1])     atomicAdd( &(concentrationOut_sh[threadIdx.x][threadIdx.y + 1]) , sendVal);      //Left 	
  if (isFree_sh[threadIdx.x + 2][threadIdx.y + 1]) atomicAdd( &(concentrationOut_sh[threadIdx.x + 2][threadIdx.y + 1]) , sendVal);  //Right
  if (isFree_sh[threadIdx.x + 1][threadIdx.y])     atomicAdd( &(concentrationOut_sh[threadIdx.x + 1][threadIdx.y]) , sendVal);      //Down
  if (isFree_sh[threadIdx.x + 1][threadIdx.y + 2]) atomicAdd( &(concentrationOut_sh[threadIdx.x + 1][threadIdx.y + 2]) , sendVal);  //Up
  __syncthreads();
//   if ( isFree_sh[threadIdx.x + 1][threadIdx.y + 1] ){
  //Write shared memory to global
  concentrationOut[tid] = concentrationOut_sh[threadIdx.x+1][threadIdx.y+1];
  //Write shared memory boundary
  //Left boundary
  if (t_j == 0)                atomicAdd( &(concentrationOut[ (nWidth-1) + t_i*nWidth ]), concentrationOut_sh[0][threadIdx.y + 1] );
  else if ( threadIdx.x == 0 ) atomicAdd( &(concentrationOut[ (t_j-1) + t_i*nWidth ]), concentrationOut_sh[0][threadIdx.y + 1] );
  //Rigth boundary
  if (t_j == nWidth-1)                    atomicAdd( &(concentrationOut[ t_i*nWidth ]), concentrationOut_sh[blockDim.x + 1][threadIdx.y + 1] );
  else if ( threadIdx.x == blockDim.x-1 ) atomicAdd( &(concentrationOut[ (t_j+1) + t_i*nWidth ]), concentrationOut_sh[blockDim.x + 1][threadIdx.y + 1] );
  //Down boundary
  if (t_i == 0)                atomicAdd( &(concentrationOut[ t_j + (nHeight-1)*nWidth ]), concentrationOut_sh[threadIdx.x + 1][0] );
  else if ( threadIdx.y == 0 ) atomicAdd( &(concentrationOut[ t_j + (t_i-1)*nWidth ]), concentrationOut_sh[threadIdx.x + 1][0] );
  //Up boundary
  if (t_i == nHeight-1)                    atomicAdd( &(concentrationOut[ t_j ]), concentrationOut_sh[threadIdx.x + 1][ blockDim.y + 1] );
  else if ( threadIdx.y == blockDim.y-1 )  atomicAdd( &(concentrationOut[ t_j + (t_i+1)*nWidth ]), concentrationOut_sh[threadIdx.x + 1][blockDim.y + 1] );
  
}*/