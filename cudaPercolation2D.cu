#include "hip/hip_runtime.h"
// #include <pycuda-complex.hpp>
// #include <surface_functions.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

typedef unsigned char uchar;

texture< int, hipTextureType2D, hipReadModeElementType> tex_isFree;
texture<float, hipTextureType2D, hipReadModeElementType> tex_concentrationIn;
/////////////////////////////////////////////////////////////////////////////////////// 
/////////////////////////////////////////////////////////////////////////////////////// 
__global__ void main_kernel_tex( const int nWidth, const int nHeight, float hx, int *isFreeAll,
			     float *concentrationOut ){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int tid = t_j + t_i*blockDim.x*gridDim.x;
   
  //Read neighbors occupancy
  int left_isFree   =  tex2D( tex_isFree, t_j-1, t_i ); ; 
  int right_isFree  =  tex2D( tex_isFree, t_j+1, t_i ); ; 
  int up_isFree     =  tex2D( tex_isFree, t_j, t_i+1 ); ; 
  int down_isFree   =  tex2D( tex_isFree, t_j, t_i-1 ); ; 
  //Set PERIODIC boundary conditions
  if (t_i == 0)           down_isFree = isFreeAll[ t_j + (nHeight-1)*nWidth ];
  if (t_i == (nHeight-1))   up_isFree = isFreeAll[ t_j ];
  if (t_j == 0)           left_isFree = isFreeAll[ (nWidth-1) + (t_i)*nWidth ];
  if (t_j == (nWidth-1)) right_isFree = isFreeAll[ (t_i)*nWidth ];

  //Read neighbors concentration
  float center_C = tex2D( tex_concentrationIn, t_j,   t_i );
  float left_C   = tex2D( tex_concentrationIn, t_j-1, t_i );
  float right_C  = tex2D( tex_concentrationIn, t_j+1, t_i );
  float up_C     = tex2D( tex_concentrationIn, t_j, t_i+1 );
  float down_C   = tex2D( tex_concentrationIn, t_j, t_i-1 );
  //Set PERIODIC boundary conditions
  if (t_i == 0)           down_C = tex2D( tex_concentrationIn, t_j, nHeight-1 );
  if (t_i == (nHeight-1))   up_C = tex2D( tex_concentrationIn, t_j, 0 );
  if (t_j == 0)           left_C = tex2D( tex_concentrationIn, nWidth-1, t_i );
  if (t_j == (nWidth-1)) right_C = tex2D( tex_concentrationIn, 0, t_i );
 
  float newConcentration = hx*left_C + (1.f - hx)*(right_C + down_C + up_C )/3.f +
      ( hx*(1 - right_isFree) + (1.f-hx)*( 3 - ( left_isFree + down_isFree + up_isFree ) )/3.f )*center_C;
      
  if ( isFreeAll[tid] )  concentrationOut[tid] = newConcentration;
}




/////////////////////////////////////////////////////////////////////////////////////// 
/////////////////////////////////////////////////////////////////////////////////////// 
__global__ void main_kernel_shared( const int nWidth, const int nHeight, cudaP hx, uchar *isFreeAll,
			          cudaP *concIn, cudaP *concentrationOut, uchar *activeBlocks ){
  const int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  const int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  const int tid = t_j + t_i*blockDim.x*gridDim.x;
  
//   __shared__ uchar activeBlock;
//   if ( threadIdx.x == 0 and threadIdx.y ==0 ) activeBlock = activeBlocks[blockIdx.x + blockIdx.y*gridDim.x ];
//   __syncthreads();
//   if ( !activeBlock ) return;
  
  //Read my neighbors concentration
  __shared__ uchar   isFree_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  __shared__ cudaP conc_sh[ %(B_WIDTH)s + 2 ][ %(B_HEIGHT)s + 2 ];
  conc_sh[threadIdx.x+1][threadIdx.y+1] =    concIn[tid] ;
  isFree_sh[threadIdx.x+1][threadIdx.y+1] = isFreeAll[tid];
  //Left boundary
  if (t_j == 0){
    conc_sh[0][threadIdx.y+1] =    concIn[ (nWidth-1) + t_i*nWidth ];
    isFree_sh[0][threadIdx.y+1] = isFreeAll[ (nWidth-1) + t_i*nWidth ];
  }
  else if ( threadIdx.x == 0 ){
    conc_sh[0][threadIdx.y+1] =    concIn[ (t_j-1) + t_i*nWidth ];
    isFree_sh[0][threadIdx.y+1] = isFreeAll[ (t_j-1) + t_i*nWidth ];
  }
  //Right boundary
  if (t_j == nWidth-1){
    conc_sh[blockDim.x+1][threadIdx.y+1] =    concIn[ t_i*nWidth ];
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFreeAll[ t_i*nWidth ];
  }
  else if ( threadIdx.x == blockDim.x-1 ){
    conc_sh[blockDim.x+1][threadIdx.y+1] =    concIn[ (t_j+1) + t_i*nWidth ];
    isFree_sh[blockDim.x+1][threadIdx.y+1] = isFreeAll[ (t_j+1) + t_i*nWidth ];
  }
  //Down boundary
  if (t_i == 0){
    conc_sh[threadIdx.x+1][0] =    concIn[ t_j + (nHeight-1)*nWidth ];
    isFree_sh[threadIdx.x+1][0] = isFreeAll[ t_j + (nHeight-1)*nWidth ];
  }
  else if ( threadIdx.y == 0 ){
    conc_sh[threadIdx.x+1][0] =    concIn[ t_j + (t_i-1)*nWidth ];
    isFree_sh[threadIdx.x+1][0] = isFreeAll[ t_j + (t_i-1)*nWidth ];
  }
  //Up boundary
  if (t_i == nHeight-1){
    conc_sh[threadIdx.x+1][blockDim.y+1] =    concIn[ t_j ];
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFreeAll[ t_j ];
  }
  else if ( threadIdx.y == blockDim.y-1 ){
    conc_sh[threadIdx.x+1][blockDim.y+1] =    concIn[ t_j + (t_i+1)*nWidth ];
    isFree_sh[threadIdx.x+1][blockDim.y+1] = isFreeAll[ t_j + (t_i+1)*nWidth ];
  }
  __syncthreads();
  	 
  cudaP oneThird = 1.0/3;

//   cudaP newConc = hx*( conc_sh[threadIdx.x][threadIdx.y+1] + ( 1 - isFree_sh[threadIdx.x+2][threadIdx.y+1] )*conc_sh[threadIdx.x+1][threadIdx.y+1] ) +
//     oneThird*( 1 - hx )*( conc_sh[threadIdx.x+2][threadIdx.y+1] + conc_sh[threadIdx.x+1][threadIdx.y] + conc_sh[threadIdx.x+1][threadIdx.y+2] +
//                          conc_sh[threadIdx.x+1][threadIdx.y+1]*( 3 - ( isFree_sh[threadIdx.x][threadIdx.y+1] + isFree_sh[threadIdx.x+1][threadIdx.y] + isFree_sh[threadIdx.x+1][threadIdx.y+2] ) ) );
    
  if ( isFree_sh[threadIdx.x+1][threadIdx.y+1] )  concentrationOut[tid] =
	   hx*( conc_sh[threadIdx.x][threadIdx.y+1] + ( 1 - isFree_sh[threadIdx.x+2][threadIdx.y+1] )*conc_sh[threadIdx.x+1][threadIdx.y+1] ) +
    oneThird*( 1 - hx )*( conc_sh[threadIdx.x+2][threadIdx.y+1] + conc_sh[threadIdx.x+1][threadIdx.y] + conc_sh[threadIdx.x+1][threadIdx.y+2] +
                         conc_sh[threadIdx.x+1][threadIdx.y+1]*( 3 - ( isFree_sh[threadIdx.x][threadIdx.y+1] + isFree_sh[threadIdx.x+1][threadIdx.y] + isFree_sh[threadIdx.x+1][threadIdx.y+2] ) ) );
    

}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void findActivity_kernel( cudaP minVal, cudaP *concentration, uchar *activeBlocks ){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int tid = t_j + t_i*blockDim.x*gridDim.x;
  int tid_b = threadIdx.x + threadIdx.y*blockDim.x;

  __shared__ cudaP concentration_sh[ %(THREADS_PER_BLOCK)s ];
  concentration_sh[tid_b] = concentration[tid];
  __syncthreads();
  
  int i = blockDim.x*blockDim.y / 2;
  while ( i > 0 ){
    if ( tid_b < i ) concentration_sh[tid_b] = concentration_sh[tid_b] + concentration_sh[tid_b+i];
    __syncthreads();
    i /= 2;
  }
  if (concentration_sh[0] >= minVal ){
    if  ( tid_b < 3 ){
      // left,  center and right
      if ( ( blockIdx.x > 0 ) and ( blockIdx.x < gridDim.x-1 ) ) activeBlocks[ blockIdx.x + (tid_b-1) + blockIdx.y*gridDim.x ] = (uchar) 1;
      // down and up
      if ( ( tid_b != 1) and (blockIdx.y > 0) and ( blockIdx.y < gridDim.y-1 ) ) activeBlocks[ blockIdx.x + (blockIdx.y+tid_b-1)*gridDim.x  ] = (uchar) 1;
//       //right 
//       if (blockIdx.x < gridDim.x-1) activeBlocks[ blockIdx.x+1 + blockIdx.y*gridDim.x ] = (uchar) 1;
//       //left
//       if (blockIdx.x > 0) activeBlocks[ (blockIdx.x-1) + blockIdx.y*gridDim.x ] = (uchar) 1;
//       if ( tid_b == 0 ){
//       //up 
// 	if (blockIdx.y < gridDim.y-1) activeBlocks[ blockIdx.x + (blockIdx.y+1)*gridDim.x  ] = (uchar) 1;
// 	//Down 
// 	if (blockIdx.y > 0) activeBlocks[ blockIdx.x + (blockIdx.y-1)*gridDim.x  ] = (uchar) 1;
//       }
    }
  }
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void getActivity_kernel(  uchar *activeBlocks, uchar *activeThreads ){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int tid = t_j + t_i*blockDim.x*gridDim.x ;
  int tid_b = threadIdx.x + threadIdx.y*blockDim.x;
  int bid = blockIdx.x + blockIdx.y*gridDim.x;
  
  __shared__ uchar activeBlock;
  if (tid_b == 0 ) activeBlock = activeBlocks[bid];
  __syncthreads();
  uchar active = 0;
  if ( activeBlock ) active = (uchar) 1;
  activeThreads[tid] = active;
}
